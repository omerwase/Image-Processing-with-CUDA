#include "hip/hip_runtime.h"
/*
 * Programming Assignment 5 for CS 6F03 Winter 2017 Term
 * Filename: pa4.c
 * By: Omer Waseem (#000470449) and Erica Cheyne  (#001201341)
 * Description:
 * 	- Image blurred based on given radius using CUDA
 */

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
extern "C" {
#include "ppmFile.h"
}

// kernel function that blurs entire image based on block/thread vector IDs
__global__ void blur(int *d_w, int *d_h, int *d_r, unsigned char *d_input, unsigned char *d_output) {
    int i, j, x, y, chan, temp, num, offset;
    i = blockIdx.x * blockDim.x + threadIdx.x;
    j = blockIdx.y * blockDim.y + threadIdx.y;
	for (chan = 0; chan < 3; chan++) {
		temp = 0;
		num = 0;
		for (y = j - (*d_r); y <= j + (*d_r); y++) {
	
			for (x = i - (*d_r); x <= i + (*d_r); x++) {
				if (x >= 0 && x < *d_w && y >= 0 && y < *d_h) {
					offset = (y * (*d_w) + x) * 3 + chan;
					temp += d_input[offset];
					num++;
				}
			}
		}
		temp /= num;
		offset = (j * (*d_w) + i) * 3 + chan;
		d_output[offset] = temp;
	}
}

int main (int argc, char *argv[]) {
    int w, h, r, temp;
    Image *inImage;
    Image *outImage;
    unsigned char *data;
    double time;
    clock_t begin, end;
	unsigned char *d_input;
    unsigned char *d_output;
    int *d_w, *d_h, *d_r;
    
    
    // check for correct number of input arguments
    if (argc != 4) {
		printf("Incorrect input arguments. Should be: <r> <input>.ppm <output>.ppm\n");
        return 0;
    }
    
	
	r = atoi(argv[1]);
	inImage = ImageRead(argv[2]);
	w = inImage->width;
	h = inImage->height;
	data = inImage->data;
	printf("Using image: %s, width: %d, height: %d, blur radius: %d\n",argv[2],w,h,r);
    printf("Waiting for GPU ...\n");

    // Grids are based on image size with blocks of 32x32
    dim3 blockD(32, 32);
    dim3 gridD((w + blockD.x - 1) / blockD.x, (h + blockD.y - 1) / blockD.y);
    
    // allocate GPU memory
    hipMalloc((void**)&d_input, sizeof(unsigned char*) * w * h * 3);
    hipMalloc((void**)&d_output, sizeof(unsigned char*) * w * h * 3);
    hipMalloc((void**)&d_w, sizeof(int*));
    hipMalloc((void**)&d_h, sizeof(int*));
    hipMalloc((void**)&d_r, sizeof(int*));
    
    // copy values to GPU
    hipMemcpy(d_input, data, w * h * 3, hipMemcpyHostToDevice);
    hipMemcpy(d_w, &w, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_h, &h, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_r, &r, sizeof(int), hipMemcpyHostToDevice);
    
	printf("Blurring image ...\n");
    printf("Grid size: %d x %d\n", gridD.x, gridD.y);
	printf("Block size: %d x %d\n", blockD.x, blockD.y);
    printf("Total number of threads: %d\n", gridD.x * gridD.y * blockD.x * blockD.y);
    
    // begin blurring time
    begin = clock();
    
    // blur image using CUDA (except top and bottom edge)
    blur<<<gridD, blockD>>>(d_w, d_h, d_r, d_input, d_output);

	// create new image for output
	outImage = ImageCreate(w, h);
	ImageClear(outImage, 255, 255, 255);
	
	hipDeviceSynchronize();
    hipDeviceSynchronize();
    
    // end blurring time
    end = clock();
    
	// copy blurred output from GPU to host
	printf("Blurring complete, assembling image ...\n");
    
    temp = w * h * 3;
    hipMemcpy(outImage->data, d_output, temp, hipMemcpyDeviceToHost);
	
	// write blurred image
	ImageWrite(outImage, argv[3]);
	printf("Blurred image created: %s\n", argv[3]);
    
    time = (double)(end-begin) / CLOCKS_PER_SEC;
    printf("Blurring execution time: %e s\n", time);
    
	free(inImage->data);
	free(outImage->data);
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_w);
    hipFree(d_h);
    hipFree(d_r);
    
    return 0;
}
